#include "hip/hip_runtime.h"
#define _CRT_SECURE_NO_WARNINGS
#define OLC_PGE_APPLICATION
#include "olcPixelGameEngine.h"
#include "utils.cpp"
#include "Image.cpp"
#include <iostream>
#include <chrono>
#include <stdlib.h>
#include <omp.h>
#include <vector>

// FILE MANATGMENT:
#include <fstream>

using namespace std;
using namespace olc;


const int height = 1080;
const int width = height * 1.7778;
int iterations = 100;

double x_bounds[2] = { -2.5, 2.5 };
double y_bounds[2] = { -1.5, 1.5 };

double x_sum;
double y_sum;

int mouseXBefore;
int mouseYBefore;

bool update = false;

float* fractal = new float[width * height];
bool drawIterations = false;

float threshold = 4;
		
int dimensions = width * height;


void PrintBounds() {
	cout << endl << "Current BOUNDS:" << endl;
	cout << "\t" << "X_MIN: " << x_bounds[0] << endl;
	cout << "\t" << "X_MAX: " << x_bounds[1] << endl;
	cout << "\t" << "Y_MIN: " << y_bounds[0] << endl;
	cout << "\t" << "Y_MAX: " << y_bounds[1] << endl;
}


vd2d MouseToFractalCoordenades(vi2d MouseCoordenades, float x_bound_min = x_bounds[0], float x_bound_max = x_bounds[1], float y_bound_min = y_bounds[0], float y_bound_max = y_bounds[1]) {
	double normalizedX = (double)MouseCoordenades.x / (double)width;
	double normalizedY = (double)MouseCoordenades.y / (double)height;

	double espacioX = absolute(x_bound_min - x_bound_max);
	double espacioY = absolute(y_bound_min - y_bound_max);

	return vd2d(x_bound_min + normalizedX * espacioX, y_bound_max - normalizedY * espacioY);
}

vi2d FractalToMouseCoordenades(vd2d FractalCoordenades, float x_bound_min, float x_bound_max, float y_bound_min, float y_bound_max) {
	return vi2d((FractalCoordenades.x - x_bound_min) * (double)width / (absolute(x_bound_min - x_bound_max)), (-FractalCoordenades.y + y_bound_max) * (double)height / (absolute(y_bound_min - y_bound_max)));
}

// ITERACIONS AQUI //
__global__
void FractalCalculator(int dimensions, float *fractal, int iter, double x_bounds_s, double x_bounds_e, double y_bounds_s, double y_bounds_e) 
{
	int index = threadIdx.x;
	int stride = blockDim.x;
	
	
	for (int i=index; i < dimensions; i+=stride) {
		fractal[i] = 0;
		
		int division = (int)((float)i / (float)height);
		int residual = i % height;
		
		int coordenades[2] = { division, residual };
		double normalizedX = (double)coordenades[0] / (double)width;
		double normalizedY = (double)coordenades[1] / (double)height;

		double espacioX = x_bounds_s - x_bounds_e;
		double espacioY = y_bounds_s - y_bounds_e;
		
		// ABSOLUTE //
		if(espacioX < 0) {
			espacioX = -espacioX;
		}
		if(espacioY < 0) {
			espacioY = -espacioY;
		}

		double x = 0;
		double y = 0;

		double cr = x_bounds_s + normalizedX * espacioX;
		double ci = y_bounds_e - normalizedY * espacioY;

		for (int k = 0; k < iter; k++)
		{
			if (x * x + y * y > 4)
			{
				fractal[i] = (float)k / (float)iter;
				break;
			}

			double x_t = x;

			x = x * x - y * y + cr;
			y = 2 * y * x_t + ci;

		}
			//x = x * x * x * x + y * y * y * y - 6 * x * x * y * y + cr;
			//y = 4 * x_t * x_t * x_t * y - 4 * x_t * y * y * y + ci;
	}
}
// ITERACIONS AQUI //

class Example : public PixelGameEngine
{
public:
	Example()
	{
		sAppName = "Conjunt de Julia";
	}

public:
	byte runningFrames;

	int level = 0;

	struct bound
	{
		double x_0, x_1, y_0, y_1;
		bound(double x0, double x1, double y0, double y1) {
			x_0 = x0;
			x_1 = x1;
			y_0 = y0;
			y_1 = y1;
		}
	};

	vector<bound> bounds;

	void EraseBounds() {
		size_t boundsSize = bounds.size() - 1;

		for (size_t i = level; i < boundsSize; i++)
		{
			bounds.pop_back();
		}
	}

	void UpdateBounds() {

		if (level > bounds.size() - 1) {
			level = bounds.size() - 1;
		}
		else if (level < 0)
		{
			level = 0;
		}

		x_bounds[0] = bounds[level].x_0;
		x_bounds[1] = bounds[level].x_1;
		y_bounds[0] = bounds[level].y_0;
		y_bounds[1] = bounds[level].y_1;
	}

	bool OnUserCreate() override
	{
		// Match boundings to 16:9 aspect-ratio
		//absolute(x_bounds[0] - x_bounds[1]) / absolute(y_bounds[0] - y_bounds[1]) = 16 / 9
		//(16/9)/ absolute(x_bounds[0] - x_bounds[1]) = 1 / absolute(y_bounds[0] - y_bounds[1])
		//(9/16)* absolute(x_bounds[0] - x_bounds[1]) = absolute(y_bounds[0] - y_bounds[1])
		// Com sabem que y_bounds[0] i y_bounds[1] han de ser inversos, y_bounds[1] = -y_bounds[0]
		//(9/16)* absolute(x_bounds[0] - x_bounds[1]) = absolute(2 * y_bounds[0])
		//(9/16)* absolute(x_bounds[0] - x_bounds[1]) / 2 = y_bounds[0]
		y_bounds[1] = (float)9 / (float)16 * absolute(x_bounds[0] - x_bounds[1]) / (float)2;
		y_bounds[0] = -y_bounds[1];

		bound starting = bound(x_bounds[0], x_bounds[1], y_bounds[0], y_bounds[1]);
		bounds.insert(bounds.end(), starting);
		UpdateBounds();

		runningFrames = 0;
		PrintBounds();
		
		
		hipMallocManaged(&fractal, dimensions*sizeof(float));
		UpdateFractal();

		return true;
	}

	vi2d startingPos;
	vi2d endingPos;


	// DRAW ITERATIONS VARIABLES //
	double currentX;
	double currentY;
	double initialX;
	double initialY;
	int iter = 0;
	// DRAW ITERATIONS VARIABLES //

	double totalTime = 0;
	bool OnUserUpdate(float fElapsedTime) override
	{
		runningFrames += 1;
		
		// INPUT //
		update = HandleInput();
		// INPUT //

		totalTime += fElapsedTime;

		// DIBUAR EL CUADRADITO ESE //
		if (updateDrawingEveryFrame && runningFrames % 2 == 0) {
			DrawScreen(false);

			vi2d rectangle[2] = {
				startingPos,
				vi2d()
			};

			switch (startingPos.x > GetMousePos().x)
			{
			case true:
				if (startingPos.y > GetMousePos().y) {
					rectangle[1] = vi2d(GetMousePos().x - startingPos.x, (float)height / (float)width * (GetMousePos().x - startingPos.x));
				}
				else {
					rectangle[1] = vi2d(GetMousePos().x - startingPos.x, (float)height / (float)width * (-GetMousePos().x + startingPos.x));
				}
				break;
			case false:
				if (startingPos.y < GetMousePos().y) {
					rectangle[1] = vi2d(GetMousePos().x - startingPos.x, (float)height / (float)width * (GetMousePos().x - startingPos.x));
				}
				else {
					rectangle[1] = vi2d(GetMousePos().x - startingPos.x, (float)height / (float)width * (-GetMousePos().x + startingPos.x));
				}
				break;
			}


			DrawRect(rectangle[0], rectangle[1], Pixel(200, 200, 200));

			return true;
		}
		// DIBUAR EL CUADRADITO ESE //

		if (update) {
			UpdateFractal();
		}
		return true;


		if (!update) return true;

		auto begin = chrono::high_resolution_clock::now();


		// ACTUALIZAR EL FRACTAL //
		UpdateFractal();
		// ACTUALIZAR EL FRACTAL //


		update = false;

		auto end = chrono::high_resolution_clock::now();
		auto elapsed = chrono::duration_cast<chrono::nanoseconds>(end - begin);

		float elapsedTime = elapsed.count() * 1e-9;
		cout << "The update elapsed: " << elapsedTime << " seconds. Currently are " << iterations << " iterations." << endl;
		if (!(lastElapsedTime == 0)) {
			float plusPercentage = 100 * elapsedTime / lastElapsedTime - 100;

			char whattosay = (plusPercentage > 0) ? '+' : '-';
			cout << whattosay;
			cout << absolute(plusPercentage) << "%." << endl;
		}
		lastElapsedTime = elapsedTime;


		PrintBounds();

		return true;
	}

private:
	float lastElapsedTime = 0;
	vi2d startingMousePosition, endingMousePosition;
	bool updateDrawingEveryFrame = false;

	void StartIterate() {
		DrawScreen(false);
		
		iter = 0;
		currentX = 0;
		currentY = 0;
		initialX = MouseToFractalCoordenades(startingMousePosition).x;
		initialY = MouseToFractalCoordenades(startingMousePosition).y;

		// Dibuixem un cercle on hem clicat.
		DrawCircle(startingMousePosition, 3, olc::RED);

		drawIterations = true;
	}


	bool HandleInput() {
		bool anyInput = false;

		if (GetKey(olc::H).bPressed) {
			// HELP!
			cout << endl;
			cout << "Animation." << endl;
			cout << "\tP: Pause or Reanude the animation. With the fractal paused you can use movement." << endl;
			cout << "Iterations edit commands." << endl;
			cout << "\tNumPad+ or S: Add 10 iterations;" << endl;
			cout << "\tNumPad- or R: Substract 10 iterations;" << endl;
			cout << "\tNumPad* or M: Multiply by 2 the iterations;" << endl;
			cout << "\tNumPad/ or D: Divide by 2 the iterations." << endl;
			cout << endl;
			cout << "Iterate points." << endl;
			cout << "\tI: Introduce number of iterations;" << endl;
			cout << endl;
			cout << "Movement." << endl;
			cout << "\tThe program is made to match the aspect-ratio of 16:9 everytime." << endl;
			cout << "\tHeld Mouse Left Button and drag it wherever you want." << endl;
			cout << "\tReleased to calculate the updated fractal." << endl;
			cout << "\tLeft-arrow: back to the previous bounds." << endl;
			cout << "\tRight-arrow: forward to the bounds." << endl;
			cout << "Other." << endl;
			cout << "\tMake an image! -> NumPad-Enter or Enter.";
			cout << endl;
			cout << "Have fun!" << endl;
			cout << endl;
		}


		// ITERATIONS //
		if (GetKey(NP_ADD).bReleased || GetKey(S).bReleased) {
			iterations += 10;
			anyInput = true;
		}
		if (GetKey(NP_SUB).bReleased || GetKey(R).bReleased) {
			iterations -= 10;
			anyInput = true;
		}
		if (GetKey(NP_MUL).bReleased || GetKey(M).bReleased) {
			iterations *= 2;
			anyInput = true;
		}
		if (GetKey(NP_DIV).bReleased || GetKey(D).bReleased) {
			iterations /= 2;
			anyInput = true;
		}

		if (GetKey(olc::LEFT).bReleased && level != 0) {
			cout << "Going backwards..." << endl;
			level--;
			UpdateBounds();
			UpdateFractal();
		}
		if(GetKey(olc::RIGHT).bReleased && level != bounds.size()-1)
		{
			cout << "Going forward..." << endl;
			level++;
			UpdateBounds();
			UpdateFractal();
		}
		if (GetKey(olc::I).bReleased) {

			int i;
			cout << "Introdueix el nomre d'iteracions:" << endl;
			cin >> i;
			iterations = i;
			anyInput = true;
		}


		// SAVE IMAGE //
		if (GetKey(olc::ENTER).bPressed) {
			int imageHeight = height;
			int imageWidth = width;
			Image fractalImage(width, height);

			// IMAGE NAME HAS TO BE UNIQUE IF WE WANT TO STORE MORE THAN ONE //
			auto end = std::chrono::system_clock::now();
			std::time_t end_time = std::chrono::system_clock::to_time_t(end);
			string date = std::ctime(&end_time);

			for (int c = 0; c < date.size(); c++)
			{
				if (date[c] == ' ')
					date[c] = '_';
				if (date[c] == ':' || date[c] == '\n')
					date[c] = '-';
			}
			string path = "Captured_On_" + date + ".bmp";
			// IMAGE NAME HAS TO BE UNIQUE IF WE WANT TO STORE MORE THAN ONE //


			/*	S'HAURÀ DE RECALCULAR EL FRACTAL SI ES VOL GUARDAR EN UNA 
				RESOLUCIÓ MÉS GRAN QUE LA QUE HI HA AL PROJECTE.

				CANVIAR TAMBÉ LA PART D'ABAIX.
			*/
			
			// SET ALL IMAGE COLORS //
			int y_f = imageHeight;  // Per revertir la imatge que es genera al revés si no.
			for (int y = 0; y < imageHeight; y++)
			{
				for (int x = 0; x < imageWidth; x++)
				{
					int index = x * imageHeight + y_f;
					fractalImage.SetColor(NewColor(fractal[index], fractal[index], fractal[index]), x, y);
				}
				y_f -= 1;
			}
			// SET ALL IMAGE COLORS //

			// SAVE THE IMAGE //
			fractalImage.Export(path.c_str());
			// SAVE THE IMAGE //
		}
		// SAVE IMAGE //


		// MOUSE INPUT //
		if (GetMouse(0).bPressed) {
			drawIterations = false;
			startingMousePosition = GetMousePos();
			cout << "Mouse Input.\n\tStarted at: " << startingMousePosition << endl;
			startingPos = GetMousePos();
			updateDrawingEveryFrame = true;
		}
		if (!GetMouse(0).bHeld) {
			updateDrawingEveryFrame = false;
		}
		if (GetMouse(0).bReleased) {

			endingMousePosition = GetMousePos();

			vi2d rectangle[2] = {
				startingMousePosition,
				vi2d(endingMousePosition.x - startingMousePosition.x, (float)height / (float)width * (endingMousePosition.x - startingMousePosition.x))
			};

			// CORRIGIENDO LA POSICIÓN FINAL //
			vi2d endedCorrectedMousePosition = vi2d();
			switch (startingMousePosition.x > endingMousePosition.x)
			{
			case true:
				if (startingMousePosition.y > endingMousePosition.y) {
					endedCorrectedMousePosition = vi2d(endingMousePosition.x, (endingMousePosition.x - startingMousePosition.x) * height / width + startingMousePosition.y);
				}
				else {
					endedCorrectedMousePosition = vi2d(endingMousePosition.x, (-endingMousePosition.x + startingMousePosition.x) * height / width + startingMousePosition.y);
				}
				break;
			case false:
				if (startingMousePosition.y < endingMousePosition.y) {
					endedCorrectedMousePosition = vi2d(endingMousePosition.x, (endingMousePosition.x - startingMousePosition.x) * height / width + startingMousePosition.y);
				}
				else {
					endedCorrectedMousePosition = vi2d(endingMousePosition.x, (-endingMousePosition.x + startingMousePosition.x) * height / width + startingMousePosition.y);
				}
				break;
			}
			// CORRIGIENDO LA POSICIÓN FINAL //

			cout << "\tEnded at : " << endedCorrectedMousePosition << endl;

			DrawRect(rectangle[0], rectangle[1], Pixel(255, 105, 105));

			vd2d fractalCoordenadesRectangle[2] = {
				MouseToFractalCoordenades(rectangle[0]),
				MouseToFractalCoordenades(endedCorrectedMousePosition)
			};


			if ((startingMousePosition - endingMousePosition).mag2() < 900) {

				StartIterate();
				return anyInput;
			}

			cout << "Moving to:" << endl;

			// No preguntes cómo. //
			switch (startingMousePosition.x > endingMousePosition.x)
			{
			case true:
				x_bounds[0] = fractalCoordenadesRectangle[1].x;
				x_bounds[1] = fractalCoordenadesRectangle[0].x;
				if (startingMousePosition.y < endingMousePosition.y) {
					y_bounds[0] = fractalCoordenadesRectangle[1].y;
					y_bounds[1] = fractalCoordenadesRectangle[0].y;
				}
				else {
					y_bounds[0] = fractalCoordenadesRectangle[0].y;
					y_bounds[1] = fractalCoordenadesRectangle[1].y;
				}
				break;
			case false:
				x_bounds[0] = fractalCoordenadesRectangle[0].x;
				x_bounds[1] = fractalCoordenadesRectangle[1].x;
				if (startingMousePosition.y < endingMousePosition.y) {
					y_bounds[0] = fractalCoordenadesRectangle[1].y;
					y_bounds[1] = fractalCoordenadesRectangle[0].y;
				}
				else {
					y_bounds[0] = fractalCoordenadesRectangle[0].y;
					y_bounds[1] = fractalCoordenadesRectangle[1].y;
				}
				break;
			}
			// No preguntes cómo. //


			// UPDATE BOUNDS:
			EraseBounds();
			bounds.insert(bounds.end(), bound(x_bounds[0], x_bounds[1], y_bounds[0], y_bounds[1]));
			level++;
			// UPDATE BOUNDS:

			cout << "\tX_MIN: " << x_bounds[0] << endl;
			cout << "\tX_MAX: " << x_bounds[1] << endl;
			cout << "\tY_MIN: " << y_bounds[0] << endl;
			cout << "\tY_MAX: " << y_bounds[1] << endl;

			anyInput = true;
		}

		return anyInput;
	}

	void UpdateFractal() {		
		auto begin = chrono::high_resolution_clock::now();
		
		x_sum = (absolute(x_bounds[0] - x_bounds[1])) / width;
		y_sum = (absolute(y_bounds[0] - y_bounds[1])) / height;
		
		FractalCalculator<<<1, 1024>>>(dimensions, fractal, iterations, x_bounds[0], x_bounds[1], y_bounds[0], y_bounds[1]);
		
		hipDeviceSynchronize();
		
		auto end = chrono::high_resolution_clock::now();
		auto elapsed = chrono::duration_cast<chrono::nanoseconds>(end - begin);
		float elapsedTime = elapsed.count() * 1e-9;
		cout << "Update elapsed: " << elapsedTime << " seconds." << endl;
		
		DrawScreen(false);
	}

	void DrawScreen(bool time) {
		chrono::steady_clock::time_point begin;
		if (time)
			begin = chrono::high_resolution_clock::now();

		//#pragma omp parallel for
		for (int x = 0; x < width; x++) {
			for (int y = 0; y < height; y++) {
				int index = height * x + y;
				float value = fractal[index];
				
				if (value == 1) {
					Draw(x, y, Pixel(0, 0, 0));
					continue;
				}
				Draw(x, y, Pixel(value * 255, value * 255, value * 255));
				
				//Draw(x, y, pix);
			}
		}

		if (time) {
			auto end = chrono::high_resolution_clock::now();
			auto elapsed = chrono::duration_cast<chrono::nanoseconds>(end - begin);
			float elapsedTime = elapsed.count() * 1e-9;
			cout << "The paralelized drawing elapsed: " << elapsedTime << " seconds." << endl;
		}
	}
};

int main()
{
	Example demo;
	if (demo.Construct(width, height, 1, 1, true))
		demo.Start();

	return 0;
}